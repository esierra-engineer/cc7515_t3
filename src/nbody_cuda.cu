//
// Created by erick on 5/8/25.
//
#include "nbody.h"
#include <hip/hip_runtime.h>
#include "utils.h"
#include <iostream>
#define DEFAULT_MASS 1e9f

/**
 * Host Function (CPU-side)
 * Copy the input data from host memory to device memory, also known as host-to-device transfer.
 * Load the GPU program and execute, caching data on-chip for performance.
 * Copy the results from device memory to host memory, also called device-to-host transfer.
 * source: https://developer.nvidia.com/blog/cuda-refresher-cuda-programming-model/

 * @param h_bodies Array of Body
 * @param kernelFilename Kernel filename
 * @param localSize Block size
 * @param n Number of bodies
 */
void simulateNBodyCUDA(Body* h_bodies, const char* kernelFilename, int localSize, int n, float dt, float* mass, float* special_mass) {
    float deref_mass = *mass * DEFAULT_MASS;
    float deref_special_mass = *special_mass * DEFAULT_MASS;

    // destination memory address pointer
    Body* d_bodies;
    // in memory size of n bodies
    size_t size = n * sizeof(Body);

    // allocate GPU memory
    hipMalloc(&d_bodies, size);

    // copy data between host and device
    hipMemcpy(d_bodies, h_bodies, size, hipMemcpyHostToDevice);

    // 1D Kernel configuration
    int threadsPerBlock = localSize;
    int blocksNeeded = (n + threadsPerBlock - 1) / threadsPerBlock;

    dim3 blockDim(threadsPerBlock);
    dim3 gridDim(blocksNeeded);

    size_t sharedMemSize = threadsPerBlock * sizeof(Body);

    // Kernel
    hipCtx_t context;
    hipFunction_t kernel = loadKernelSource(kernelFilename, &context);

    // Kernel args deben ser punteros a los datos
    void* kernelArgs[] = {
        (void*) &d_bodies,
        (void*) &n,
        (void*) &dt,
        (void*) &deref_mass,
        (void*) &deref_special_mass
    };

    checkCudaErrors(
        hipModuleLaunchKernel(
            kernel,
            gridDim.x, 1, 1,                    // grid
            blockDim.x, 1, 1,             // block
            sharedMemSize, nullptr,                        // shared memory and stream
            kernelArgs, nullptr)                // args
        );

    // wait
    hipDeviceSynchronize();

    // retrieve the updated positions and velocities
    hipMemcpy(h_bodies, d_bodies, size, hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_bodies);

    hipCtxDestroy(context);
}